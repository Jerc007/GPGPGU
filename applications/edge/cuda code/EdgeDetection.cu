#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#define DIM1 16
#define DIM2 16


typedef struct {
	uint8_t R;
	uint8_t G;
	uint8_t B;
} pixel;					


__device__ uint8_t img_grayscaled[DIM1][DIM2]; // intermediate grayscaled image	
// IMPORTANT NOTE: the compiler takes into account that the address of this global variable (0x400) is stored in the constant memory at address c [0xe] [0x0]. 
// This zone is unknown, so to avoid problems, after compilation all the instructions containing that address in the SASS are to be changed to c [0x1] [0x6]. 		


// Edge Detection kernel
__global__ void EdgeDetection(pixel imgin[][DIM2], uint8_t imgout[][DIM2], size_t M, size_t N) 
{
	// MAINTAIN THIS PART
	int a = blockDim.x;
	int b = blockDim.y;
	int w = blockDim.z;
	int A = gridDim.x;
	int B = gridDim.y;
	int x = threadIdx.x;
	int y = threadIdx.y;
	int z = threadIdx.z;
	int X = blockIdx.x;
	int Y = blockIdx.y;
	int ops = x + (y*a) + (z*a*b) + (X*a*b*w) + (Y*a*b*w*A);
	// MAINTAIN THIS PART

	// image is MxN pixels, divided into as many subimages as the threads are
	// each thread takes as many pixels as it can in a single row
	// if pixels per thread (ppt) number is lower than pixels in a line (N), then more than one thread occupies a line, and each subimage has m=1 and n=N/tpl (threads per line)
	// if ppt is equal to N, then just one thread occupies a line, and each subimage has m=1 and n=N
	// if ppt is greater than N, then more than 1 line is assigned to a single thread, and each subimage has m=M/lpt (lines per thread) and n=N  
	//  ____________________________________
	// |_________________|__________________| ^
	// |_________________|__________________| |
	// |_________________|__________________| M
	// |_________________|__________________| |	
	// |_________________|__________________| ^	
	// |_________________|__________________|
	// 				  <- N ->	
	
	int m; // height of subimage
	int n; // width of subimage
	int r, c; // starting pixel coordinates
	int i, j; // cursors for moving through the pixels
	int16_t s; // temporary accumulator for computation
	int totPix = M*N; // total number of pixels
	int totThreads = a*b*w*A*B; // total number of threads
	int ppt; // pixels-per-thread
	int tpl; // threads-per-line (not used if ppt > N)
	int lpt; // lines-per-thread (not used if ppt <= N)

	// unfortunately division is not supported by FlexGrip architecture, but since divisions are between powers of 2, we can use the logarithm method 
	// ppt computation
	int k1 = totPix;
	int k2 = totThreads;
	int lg1 = 0;
	int lg2 = 0;
	while(k1 > 0) {
     	k1 = k1 >> 1;
    	lg1++;
    }
 	lg1--;
 	while(k2 > 0) {
 		k2 = k2 >> 1;
 		lg2++;
 	}
 	lg2--;
	ppt = 1 << (lg1 - lg2);

	if(ppt <= N) {
		// tpl computation
		k1 = N;
		k2 = ppt;
		lg1 = 0;
		lg2 = 0;
		while(k1 > 0) {
		 	k1 = k1 >> 1;
			lg1++;
		}
		lg1--;
		while(k2 > 0) {
			k2 = k2 >> 1;
			lg2++;
		}
		lg2--;
		tpl = 1 << (lg1 - lg2);
		// starting row index is found as ops shifted right of the logarithm of the number of threads contained in a line
		k1 = tpl;
		lg1 = 0;
		while(k1 > 0) {
			k1 = k1 >> 1;
			lg1++;
		}
		lg1--;
		r = ops >> lg1;
		// starting column index is found as ppt*(ops MOD tpl), i.e., ppt*(ops AND (tpl-1))
		c = ppt*(ops & (tpl-1));
		// number of rows of the subimage is just 1
		m = 1;
		// number of columns of the subimage is N/tpl (possibly all columns if ppt == N)
		k1 = N;
		k2 = tpl;
		lg1 = 0;
		lg2 = 0;
		while(k1 > 0) {
		 	k1 = k1 >> 1;
			lg1++;
		}
		lg1--;
		while(k2 > 0) {
			k2 = k2 >> 1;
			lg2++;
		}
		lg2--;
		n = 1 << (lg1 - lg2);
	}
	else {
		// lpt computation
		k1 = ppt;
		k2 = N;
		lg1 = 0;
		lg2 = 0;
		while(k1 > 0) {
		 	k1 = k1 >> 1;
			lg1++;
		}
		lg1--;
		while(k2 > 0) {
			k2 = k2 >> 1;
			lg2++;
		}
		lg2--;
		lpt = 1 << (lg1 - lg2);
		// starting row index is found as ops shifted left of the logarithm of lpt
		k1 = lpt;
		lg1 = 0;
		while(k1 > 0) {
			k1 = k1 >> 1;
			lg1++;
		}
		lg1--;
		r = ops << lg1;
		// starting column index is just 0
		c = 0;
		// number of rows of subimage is given by M/lpt
		k1 = M;
		k2 = lpt;
		lg1 = 0;
		lg2 = 0;
		while(k1 > 0) {
		 	k1 = k1 >> 1;
			lg1++;
		}
		lg1--;
		while(k2 > 0) {
			k2 = k2 >> 1;
			lg2++;
		}
		lg2--;
		m = 1 << (lg1 - lg2);
		// number of columns of subimage all the columns, so N
		n = N;
	}

	// transformation of the image in grayscale by applying Y = 0.2989*R + 0.5870*G + 0.1140*B for each pixel
	// a mixed multiplication/shifting is perfomed substitutes the division
	// multiplying for 0.2989 means multiplying for 19588/65536, i.e. shifting right of 16 positions
	// multiplying for 0.5870 means multiplying for 38469/65536, i.e. shifting right of 16 positions
	// multiplying for 0.1140 means multiplying for 7471/65536, i.e. shifting right of 16 positions
	for(i=0; i<m; i++) {
		for(j=0; j<n; j++) {
			s = ((imgin[r+i][c+j].R*0x4C84)>>16) + ((imgin[r+i][c+j].G*0x9645)>>16) + ((imgin[r+i][c+j].B*0x12DF)>>16);
			if(s > 255) img_grayscaled[r+i][c+j] = 255; 
			else img_grayscaled[r+i][c+j] = (uint8_t)s;
		}
	}

	// Then we should compute a matrix convolution between the grayscaled image and a filter which highlights edges (* indicates a convolution):
	//      			    	   0   1   0        	 
	// 		A    	   *    	   1  -4   1          
	//      			    	   0   1   0        	 
	// input matrix    *    edge detector filter      
	
	for(i=0; i<m; i++) {
		for(j=0; j<n; j++) {
			// clockwise sense
			s = 0;
			s = s + (img_grayscaled[r+i][c+j])*-4;
			if(c+j+1 <= N-1) s = s + img_grayscaled[r+i][c+j+1];
			if(r+i+1 <= M-1) s = s + img_grayscaled[r+i+1][c+j];
			if(c+j-1 >= 0) s = s + img_grayscaled[r+i][c+j-1];
			if(r+i-1 >= 0) s = s + img_grayscaled[r+i-1][c+j];
			if(s < 0) s = 0;
			else if(s > 255) s = 255;
			imgout[r+i][c+j] = (uint8_t)s;
		}
	}

}



int main(void)
{

	int blocksPerGrid = 2;
	int threadsPerBlock = 32;

	pixel input_image[DIM1][DIM2];
	uint8_t output_image[DIM1][DIM2];

	// please make sure that the number of thread is at most equal to the number of pixels, not more 
	EdgeDetection<<<blocksPerGrid, threadsPerBlock>>>(input_image, output_image, DIM1, DIM2);								

	return 0;	

}